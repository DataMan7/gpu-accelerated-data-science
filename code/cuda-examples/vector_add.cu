#include <iostream>
#include <hip/hip_runtime.h>

// CUDA kernel for vector addition
__global__ void vectorAdd(float* a, float* b, float* c, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        c[i] = a[i] + b[i];
    }
}

// CPU version for comparison
void vectorAddCPU(float* a, float* b, float* c, int n) {
    for (int i = 0; i < n; i++) {
        c[i] = a[i] + b[i];
    }
}

int main() {
    int n = 1000000;  // 1M elements
    size_t size = n * sizeof(float);

    // Allocate host memory
    float* h_a = (float*)malloc(size);
    float* h_b = (float*)malloc(size);
    float* h_c = (float*)malloc(size);
    float* h_c_cpu = (float*)malloc(size);

    // Initialize host arrays
    for (int i = 0; i < n; i++) {
        h_a[i] = rand() / (float)RAND_MAX;
        h_b[i] = rand() / (float)RAND_MAX;
    }

    // Allocate device memory
    float* d_a, *d_b, *d_c;
    hipMalloc(&d_a, size);
    hipMalloc(&d_b, size);
    hipMalloc(&d_c, size);

    // Copy data to device
    hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);

    // Launch kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_c, n);

    // Copy result back to host
    hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost);

    // CPU computation for verification
    vectorAddCPU(h_a, h_b, h_c_cpu, n);

    // Verify results
    bool success = true;
    for (int i = 0; i < n; i++) {
        if (fabs(h_c[i] - h_c_cpu[i]) > 1e-5) {
            success = false;
            break;
        }
    }

    if (success) {
        std::cout << "GPU computation successful!" << std::endl;
    } else {
        std::cout << "GPU computation failed!" << std::endl;
    }

    // Free memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    free(h_a);
    free(h_b);
    free(h_c);
    free(h_c_cpu);

    return 0;
}